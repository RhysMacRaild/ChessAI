
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>


class Board {
private:
    int turn = 0;
    int movesChecked = 0;
    Board* possibleMoves[1000];
    Board* gameHistory[300];

    //Move a piece and create a new board
    void movePiece(Board* currentBoard, int initialRow, int initialCol, int movedTooRow, int movedTooCol) {
        for (int row = 0; row < 8; row++) {
            for (int col = 0; col < 8; col++) {
                //Remove the piece from its inital square
                if (row == initialRow && col == initialCol) {
                    board[row][col] = 0;
                }
                //Move the piece to new square
                else if (row == movedTooRow && col == movedTooCol) {
                    board[row][col] = currentBoard->board[initialRow][initialCol];
                }

                //Copy the rest of the board
                else {
                    board[row][col] = currentBoard->board[row][col];
                    //(board[row]) + col = (currentBoard->board[row]) + col;
                }
            }
        }
    }

    bool moveCheck(int initRow, int initCol, int newRow, int newCol, bool white) {
        //Toggle between checking valid white and black moves
        int lowerBound;
        int upperBound;
        if (!white) {
            lowerBound = 1;
            upperBound = 6;
        }
        else {
            lowerBound = 7;
            upperBound = 12;
        }

        if (newCol >= 0 && newCol < 8) {
            if (board[newRow][newCol] == 0) {
                Board* move1 = new Board(this, initRow, initCol, newRow, newCol);
                return true;
            }
            else if (board[newRow][newCol] >= lowerBound && board[newRow][newCol] <= upperBound) {
                Board* move1 = new Board(this, initRow, initCol, newRow, newCol);
                return false;
            }

            else {
                return false;
            }
        }
    }

    void moveRook(int row, int col, bool white) {
        bool up = true;
        bool down = true;
        bool left = true;
        bool right = true;

        //Check Right
        for (int newCol = col + 1; newCol < 8; newCol++) {
            if (right) {
                right = moveCheck(row, col, row, newCol, white);
            }
            else {
                break;
            }
        }
        //Check Left
        for (int newCol = col - 1; newCol >= 0; newCol--) {
            if (left) {
                left = moveCheck(row, col, row, newCol, white);
            }
            else {
                break;
            }
        }

        //Check Down
        for (int newRow = row - 1; newRow >= 0; newRow--) {
            if (down) {
                down = moveCheck(row, col, newRow, col, white);
            }
            else {
                break;
            }
        }

        //Check Up
        for (int newRow = row + 1; newRow < 8; newRow++) {
            if (up) {
                up = moveCheck(row, col, newRow, col, white);
            }
            else {
                break;
            }
        }
    }

    void moveBishop(int row, int col, bool white) {
        bool upRight = true;
        bool upLeft = true;
        bool downRight = true;
        bool downLeft = true;

        //Check rows above current
        for (int newRow = row + 1; newRow < 8; newRow++) {

            int moveUpRightCol = col + (newRow - row);
            int moveUpLeftCol = col - (newRow - row);

            //UpRight
            if (upRight) {
                upRight = moveCheck(row, col, newRow, moveUpRightCol, white);
            }

            if (upLeft) {
                upLeft = moveCheck(row, col, newRow, moveUpLeftCol, white);
            }

        }

        //Check rows below current
        for (int newRow = row - 1; newRow >= 0; newRow--) {

            int moveDownRightCol = col + (row - newRow);
            int moveDownLeftCol = col - (row - newRow);

            //UpRight
            if (downRight) {
                downRight = moveCheck(row, col, newRow, moveDownRightCol, white);
            }

            if (downLeft) {
                downLeft = moveCheck(row, col, newRow, moveDownLeftCol, white);
            }

        }
    }


public:
    int board[8][8] = {
        {1,2,3,4,5,3,2,1},
        {6,6,6,6,6,6,6,6},
        {0,0,0,0,0,0,0,0},
        {0,0,0,0,0,0,0,0},
        {0,0,0,0,0,0,0,0},
        {7,8,9,10,11,0,0,0},
        {12,12,12,12,12,12,12,12},
        {7,8,9,10,11,9,8,7} };

    Board() {
        printBoard();
    }

    //Validate move and generate new board if valid
    Board(Board* currentBoard, int initialRow, int initialCol, int movedTooRow, int movedTooCol) {
        //Check if proposed move exists
        if ((movedTooRow >= 8 || movedTooRow < 0) || (movedTooCol >= 8 || movedTooCol < 0)) {
            delete this;
        }
        else {
            //Check if move is possible for white
            if (currentBoard->board[initialRow][initialCol] < 7) {
                //Check if a white piece is trying to move to the position of another white piece
                if (currentBoard->board[movedTooRow][movedTooCol] < 7 && currentBoard->board[movedTooRow][movedTooCol] > 0) {
                    delete this;
                }
                else {
                    movePiece(currentBoard, initialRow, initialCol, movedTooRow, movedTooCol);
                    currentBoard->addBoardToPossible(this);
                }
            }
            //Check for black
            else {
                //Check if black piece if moving to the position of another black piece
                if (currentBoard->board[movedTooRow][movedTooCol] >= 7 && currentBoard->board[movedTooRow][movedTooCol] <= 12) {
                    delete this;
                }
                else {
                    movePiece(currentBoard, initialRow, initialCol, movedTooRow, movedTooCol);
                    currentBoard->addBoardToPossible(this);
                }
            }
        }
    }

    void getMoves(bool white) {
        //White Moves
        if (white) {
            for (int row = 0; row < 8; row++) {
                for (int col = 0; col < 8; col++) {
                    //Pawn
                    if (getPositionAsChar(board[row][col]) == 'P' && row == 1 && board[row + 1][col] == 0) {
                        Board* move1 = new Board(this, row, col, row + 1, col);
                        Board* move2 = new Board(this, row, col, row + 2, col);
                    }
                    else if (getPositionAsChar(board[row][col]) == 'P') {
                        Board* move1 = new Board(this, row, col, row + 1, col);
                    }

                    //Knight
                    else if (getPositionAsChar(board[row][col]) == 'N') {
                        moveCheck(row, col, row + 2, col - 1, white );
                        moveCheck(row, col, row + 2, col + 1, white );
                        moveCheck(row, col, row - 1, col + 2, white );
                        moveCheck(row, col, row + 1, col + 2, white );
                        moveCheck(row, col, row - 2, col - 1, white );
                        moveCheck(row, col, row - 2, col + 1, white );
                        moveCheck(row, col, row - 1, col - 2, white );
                        moveCheck(row, col, row + 1, col - 2, white );
                    }

                    //Bishop
                    else if (getPositionAsChar(board[row][col]) == 'B') {
                        moveBishop(row, col, white);
                    }

                    //Rook
                    else if (getPositionAsChar(board[row][col]) == 'R') {
                        moveRook(row, col, white);
                    }

                    //Queen
                    else if (getPositionAsChar(board[row][col]) == 'Q') {
                        moveRook(row, col, white);
                        moveBishop(row, col, white);
                    }

                    //King
                    else if (getPositionAsChar(board[row][col]) == 'K') {
                        moveCheck(row, col, row + 1, col, white);
                        moveCheck(row, col, row + 1, col, white);
                        moveCheck(row, col, row - 1, col, white);
                        moveCheck(row, col, row, col+1, white);
                        moveCheck(row, col, row, col-1, white);
                        moveCheck(row, col, row + 1, col+1, white);
                        moveCheck(row, col, row - 1, col+1, white);
                        moveCheck(row, col, row + 1, col - 1, white);
                        moveCheck(row, col, row - 1, col - 1, white);
                    }
                }

            }
        }
            //Black moves
        else {
            for (int row = 0; row < 8; row++) {
                for (int col = 0; col < 8; col++) {
                    //Pawn
                    if (getPositionAsChar(board[row][col]) == 'p' && row == 6 && board[row - 1][col] == 0) {
                        Board* move1 = new Board(this, row, col, row - 1, col);
                        Board* move2 = new Board(this, row, col, row - 2, col);
                    }
                    else if (getPositionAsChar(board[row][col]) == 'p') {
                        Board* move1 = new Board(this, row, col, row - 1, col);
                    }

                    //Knight
                    else if (getPositionAsChar(board[row][col]) == 'n') {
                        moveCheck(row, col, row + 2, col - 1, white);
                        moveCheck(row, col, row + 2, col + 1, white);
                        moveCheck(row, col, row - 1, col + 2, white);
                        moveCheck(row, col, row + 1, col + 2, white);
                        moveCheck(row, col, row - 2, col - 1, white);
                        moveCheck(row, col, row - 2, col + 1, white);
                        moveCheck(row, col, row - 1, col - 2, white);
                        moveCheck(row, col, row + 1, col - 2, white);
                    }

                    //Bishop
                    else if (getPositionAsChar(board[row][col]) == 'b') {
                        moveBishop(row, col, white);
                    }

                    //Rook
                    else if (getPositionAsChar(board[row][col]) == 'r') {
                        moveRook(row, col, white);
                    }

                    //Queen
                    else if (getPositionAsChar(board[row][col]) == 'q') {
                        moveRook(row, col, white);
                        moveBishop(row, col, white);
                    }

                    //King
                    else if (getPositionAsChar(board[row][col]) == 'k') {
                        moveCheck(row, col, row + 1, col, white);
                        moveCheck(row, col, row + 1, col, white);
                        moveCheck(row, col, row - 1, col, white);
                        moveCheck(row, col, row, col + 1, white);
                        moveCheck(row, col, row, col - 1, white);
                        moveCheck(row, col, row + 1, col + 1, white);
                        moveCheck(row, col, row - 1, col + 1, white);
                        moveCheck(row, col, row + 1, col - 1, white);
                        moveCheck(row, col, row - 1, col - 1, white);
                    }
                }
            }
        }
        
    }

        void addBoardToPossible(Board * possibleBoard) {
            for (int row = 0; row < 8; row++) {
                for (int col = 0; col < 8; col++) {
                    this->possibleMoves[movesChecked] = possibleBoard;
                }
            }
            movesChecked++;
        }

        void submitTurn(Board * newBoard) {
            for (int row = 0; row < 8; row++) {
                for (int col = 0; col < 8; col++) {
                    this->gameHistory[turn]->board[row][col] = this->board[row][col];
                    this->board[row][col] = newBoard->board[row][col];
                }
            }
            turn++;
        }

        char getPositionAsChar(int positionNumber) {
            if (positionNumber == 0) {
                return ' ';
            }
            else if (positionNumber == 1) {
                return 'R';
            }
            else if (positionNumber == 2) {
                return 'N';
            }
            else if (positionNumber == 3) {
                return 'B';
            }
            else if (positionNumber == 4) {
                return 'Q';
            }
            else if (positionNumber == 5) {
                return 'K';
            }
            else if (positionNumber == 6) {
                return 'P';
            }
            else if (positionNumber == 7) {
                return 'r';
            }
            else if (positionNumber == 8) {
                return 'n';
            }
            else if (positionNumber == 9) {
                return 'b';
            }
            else if (positionNumber == 10) {
                return 'q';
            }
            else if (positionNumber == 11) {
                return 'k';
            }
            else if (positionNumber == 12) {
                return 'p';
            }
            else {
                return '!';
            }
        }

        void printBoard() {
            int tmp;
            char output[17];
            output[16] = '\0';
            for (int row = 7; row >= 0; row--) {
                for (int col = 0; col < 16; col += 2) {
                    tmp = board[row][col / 2];
                    output[col] = getPositionAsChar(tmp);
                    output[col + 1] = '|';
                }
                printf("%s", output);
                printf("\n----------------\n");
            }
            printf("\n\n");
        }

        void printAllPossibleBoards() {
            for (int boardNo = 0; boardNo < movesChecked; boardNo++) {
                possibleMoves[boardNo]->printBoard();
            }
        }
    };

    int main()
    {
        Board* board = new Board();
        board->getMoves(true);
        board->printAllPossibleBoards();

        return 0;
    }
