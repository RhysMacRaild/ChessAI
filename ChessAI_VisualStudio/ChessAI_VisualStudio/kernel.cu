
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include "board.h";
#include "layer.h";
#include "matrix.h";


class FullyConnectedLayer : public Layer {
public:
    FullyConnectedLayer(int neurons) : Layer(neurons){}
};

class NeuralNetwork {
private:
    int numberOfLayers = 0;
    Layer** layersArray = (Layer**)malloc(sizeof(Layer));


public:
    void addLayer(Layer* newLayer) {
        if (numberOfLayers == 0) {\
            numberOfLayers++;
            layersArray[0] = newLayer;
        }
        else {
            //Resize the layers array
            Layer** tmpArray = (Layer**)malloc(sizeof(Layer)*numberOfLayers);
            for (int i = 0; i < numberOfLayers; i++) {
                tmpArray[i] = layersArray[i];
            }

            free(layersArray);
            numberOfLayers++;
            layersArray = (Layer**)malloc(sizeof(layersArray) * numberOfLayers);

            for (int i = 0; i < numberOfLayers-1; i++) {
                    layersArray[i] = tmpArray[i];
            }

            //Add new layer
            layersArray[numberOfLayers-1] = newLayer;

            //Generate weights for previous layer
            layersArray[numberOfLayers - 2]->generateArrays(newLayer->getNeuronCount());
        }
    }

    void compute(int* arr) {

    }

    void printLayerWeights(int layerNumber) {
        this->layersArray[layerNumber]->printWeightsArray();
    }

    void printLayerBias(int layerNumber) {
        this->layersArray[layerNumber]->printBiasArray();
    }
};

int main()
{

    //Board* board = new Board();
    //board->getMoves(true);
    //board->printAllPossibleBoards();
    //NeuralNetwork* test = new NeuralNetwork();
    //test->addLayer(new FullyConnectedLayer(5));
    //test->addLayer(new FullyConnectedLayer(10));
    //test->printLayerBias(0);
    //test->printLayerWeights(0);
    float a[2][2] = { {1.0,2.0},{3.0,4.0} };
    float b[2][3] = { {5.0,7.0,9.0},{6.0,8.0,10.0} };
    
    float** a1;
    

    Matrix* A = new Matrix(a, 2, 2);
    Matrix* B = new Matrix(b 2, 3);


    return 0;
}
