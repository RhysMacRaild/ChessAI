
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>


    class Board {
private:
    int turn = 0;
    int movesChecked = 0;
    Board* possibleMoves[500];
    Board* gameHistory[300];

    //Move a piece and create a new board
    void movePiece(Board* currentBoard, int initialRow, int initialCol, int movedTooRow, int movedTooCol) {
        for (int row = 0; row < 8; row++) {
            for (int col = 0; col < 8; col++) {
                //Remove the piece from its inital square
                if (row == initialRow && col == initialCol) {
                    board[row][col] = 0;
                }
                //Move the piece to new square
                else if (row == movedTooRow && col == movedTooCol) {
                    board[row][col] = currentBoard->board[initialRow][initialCol];
                }

                //Copy the rest of the board
                else {
                    board[row][col] = currentBoard->board[row][col];
                    //(board[row]) + col = (currentBoard->board[row]) + col;
                }
            }
        }
    }

    bool moveCheck(int initRow, int initCol, int newRow, int newCol, bool white) {
        //Toggle between checking valid white and black moves
        int lowerBound;
        int upperBound;
        if (!white) {
            lowerBound = 1;
            upperBound = 6;
        }
        else {
            lowerBound = 7;
            upperBound = 12;
        }

        if (newCol >= 0 && newCol < 8) {
            if (board[newRow][newCol] == 0) {
                Board* move1 = new Board(this, initRow, initCol, newRow, newCol);
                return true;
            }
            else if (board[newRow][newCol] >= lowerBound && board[newRow][newCol] <= upperBound) {
                Board* move1 = new Board(this, initRow, initCol, newRow, newCol);
                return false;
            }

            else {
                return false;
            }
        }
    }

    void moveRook(int row, int col, bool white) {
        bool up = true;
        bool down = true;
        bool left = true;
        bool right = true;

        //Check Right
        for (int newCol = col + 1; newCol < 8; newCol++) {
            if (right) {
                right = moveCheck(row, col, row, newCol, white);
            }
            else {
                break;
            }
        }
        //Check Left
        for (int newCol = col - 1; newCol >= 0; newCol--) {
            if (left) {
                left = moveCheck(row, col, row, newCol, white);
            }
            else {
                break;
            }
        }

        //Check Down
        for (int newRow = row - 1; newRow >= 0; newRow--) {
            if (down) {
                down = moveCheck(row, col, newRow, col, white);
            }
            else {
                break;
            }
        }

        //Check Up
        for (int newRow = row + 1; newRow < 8; newRow++) {
            if (up) {
                up = moveCheck(row, col, newRow, col, white);
            }
            else {
                break;
            }
        }
    }

    void moveBishop(int row, int col, bool white) {
        bool upRight = true;
        bool upLeft = true;
        bool downRight = true;
        bool downLeft = true;

        //Check rows above current
        for (int newRow = row + 1; newRow < 8; newRow++) {

            int moveUpRightCol = col + (newRow - row);
            int moveUpLeftCol = col - (newRow - row);

            //UpRight
            if (upRight) {
                upRight = moveCheck(row, col, newRow, moveUpRightCol, white);
            }

            if (upLeft) {
                upLeft = moveCheck(row, col, newRow, moveUpLeftCol, white);
            }

        }

        //Check rows below current
        for (int newRow = row - 1; newRow >= 0; newRow--) {

            int moveDownRightCol = col + (row - newRow);
            int moveDownLeftCol = col - (row - newRow);

            //UpRight
            if (downRight) {
                downRight = moveCheck(row, col, newRow, moveDownRightCol, white);
            }

            if (downLeft) {
                downLeft = moveCheck(row, col, newRow, moveDownLeftCol, white);
            }

        }
    }


public:
    int board[8][8] = {
        {1,2,3,4,5,3,2,1},
        {6,6,6,6,6,6,6,6},
        {0,0,0,0,0,0,0,0},
        {0,0,0,0,0,0,0,0},
        {0,0,0,0,0,0,0,0},
        {0,0,0,0,0,0,0,0},
        {12,12,12,12,12,12,12,12},
        {7,8,9,10,11,9,8,7} };

    Board() {
        printBoard();
    }

    //Validate move and generate new board if valid
    Board(Board* currentBoard, int initialRow, int initialCol, int movedTooRow, int movedTooCol) {
        //Check if proposed move exists
        if ((movedTooRow >= 8 || movedTooRow < 0) || (movedTooCol >= 8 || movedTooCol < 0)) {
            delete this;
        }
        else {
            //Check if move is possible for white
            if (currentBoard->board[initialRow][initialCol] < 7) {
                //Check if a white piece is trying to move to the position of another white piece
                if (currentBoard->board[movedTooRow][movedTooCol] < 7 && currentBoard->board[movedTooRow][movedTooCol] > 0) {
                    delete this;
                }
                else {
                    movePiece(currentBoard, initialRow, initialCol, movedTooRow, movedTooCol);
                    currentBoard->addBoardToPossible(this);
                }
            }
            //Check for black
            else {
                //Check if black piece if moving to the position of another black piece
                if (currentBoard->board[movedTooRow][movedTooCol] >= 7 && currentBoard->board[movedTooRow][movedTooCol] <= 12) {
                    delete this;
                }
                else {
                    movePiece(currentBoard, initialRow, initialCol, movedTooRow, movedTooCol);
                    currentBoard->addBoardToPossible(this);
                }
            }
        }
    }

    void getMoves(bool white) {
        //White Moves
        if (white) {
            for (int row = 0; row < 8; row++) {
                for (int col = 0; col < 8; col++) {
                    //Pawn
                    if (getPositionAsChar(board[row][col]) == 'P' && row == 1 && board[row + 1][col] == 0) {
                        Board* move1 = new Board(this, row, col, row + 1, col);
                        Board* move2 = new Board(this, row, col, row + 2, col);
                    }
                    else if (getPositionAsChar(board[row][col]) == 'P') {
                        Board* move1 = new Board(this, row, col, row + 1, col);
                    }

                    //Knight
                    else if (getPositionAsChar(board[row][col]) == 'N') {
                        moveCheck(row, col, row + 2, col - 1, white );
                        moveCheck(row, col, row + 2, col + 1, white );
                        moveCheck(row, col, row - 1, col + 2, white );
                        moveCheck(row, col, row + 1, col + 2, white );
                        moveCheck(row, col, row - 2, col - 1, white );
                        moveCheck(row, col, row - 2, col + 1, white );
                        moveCheck(row, col, row - 1, col - 2, white );
                        moveCheck(row, col, row + 1, col - 2, white );
                    }

                    //Bishop
                    else if (getPositionAsChar(board[row][col]) == 'B') {
                        moveBishop(row, col, white);
                    }

                    //Rook
                    else if (getPositionAsChar(board[row][col]) == 'R') {
                        moveRook(row, col, white);
                    }

                    //Queen
                    else if (getPositionAsChar(board[row][col]) == 'Q') {
                        moveRook(row, col, white);
                        moveBishop(row, col, white);
                    }

                    //King
                    else if (getPositionAsChar(board[row][col]) == 'K') {
                        moveCheck(row, col, row + 1, col, white);
                        moveCheck(row, col, row + 1, col, white);
                        moveCheck(row, col, row - 1, col, white);
                        moveCheck(row, col, row, col+1, white);
                        moveCheck(row, col, row, col-1, white);
                        moveCheck(row, col, row + 1, col+1, white);
                        moveCheck(row, col, row - 1, col+1, white);
                        moveCheck(row, col, row + 1, col - 1, white);
                        moveCheck(row, col, row - 1, col - 1, white);
                    }
                }

            }
        }
            //Black moves
        else {
            for (int row = 0; row < 8; row++) {
                for (int col = 0; col < 8; col++) {
                    //Pawn
                    if (getPositionAsChar(board[row][col]) == 'p' && row == 6 && board[row - 1][col] == 0) {
                        Board* move1 = new Board(this, row, col, row - 1, col);
                        Board* move2 = new Board(this, row, col, row - 2, col);
                    }
                    else if (getPositionAsChar(board[row][col]) == 'p') {
                        Board* move1 = new Board(this, row, col, row - 1, col);
                    }

                    //Knight
                    else if (getPositionAsChar(board[row][col]) == 'n') {
                        moveCheck(row, col, row + 2, col - 1, white);
                        moveCheck(row, col, row + 2, col + 1, white);
                        moveCheck(row, col, row - 1, col + 2, white);
                        moveCheck(row, col, row + 1, col + 2, white);
                        moveCheck(row, col, row - 2, col - 1, white);
                        moveCheck(row, col, row - 2, col + 1, white);
                        moveCheck(row, col, row - 1, col - 2, white);
                        moveCheck(row, col, row + 1, col - 2, white);
                    }

                    //Bishop
                    else if (getPositionAsChar(board[row][col]) == 'b') {
                        moveBishop(row, col, white);
                    }

                    //Rook
                    else if (getPositionAsChar(board[row][col]) == 'r') {
                        moveRook(row, col, white);
                    }

                    //Queen
                    else if (getPositionAsChar(board[row][col]) == 'q') {
                        moveRook(row, col, white);
                        moveBishop(row, col, white);
                    }

                    //King
                    else if (getPositionAsChar(board[row][col]) == 'k') {
                        moveCheck(row, col, row + 1, col, white);
                        moveCheck(row, col, row + 1, col, white);
                        moveCheck(row, col, row - 1, col, white);
                        moveCheck(row, col, row, col + 1, white);
                        moveCheck(row, col, row, col - 1, white);
                        moveCheck(row, col, row + 1, col + 1, white);
                        moveCheck(row, col, row - 1, col + 1, white);
                        moveCheck(row, col, row + 1, col - 1, white);
                        moveCheck(row, col, row - 1, col - 1, white);
                    }
                }
            }
        }
        
    }

        void addBoardToPossible(Board * possibleBoard) {
            for (int row = 0; row < 8; row++) {
                for (int col = 0; col < 8; col++) {
                    this->possibleMoves[movesChecked] = possibleBoard;
                }
            }
            movesChecked++;
        }

        void submitTurn(Board * newBoard) {
            for (int row = 0; row < 8; row++) {
                for (int col = 0; col < 8; col++) {
                    this->gameHistory[turn]->board[row][col] = this->board[row][col];
                    this->board[row][col] = newBoard->board[row][col];
                }
            }
            turn++;
        }

        char getPositionAsChar(int positionNumber) {
            if (positionNumber == 0) {
                return ' ';
            }
            else if (positionNumber == 1) {
                return 'R';
            }
            else if (positionNumber == 2) {
                return 'N';
            }
            else if (positionNumber == 3) {
                return 'B';
            }
            else if (positionNumber == 4) {
                return 'Q';
            }
            else if (positionNumber == 5) {
                return 'K';
            }
            else if (positionNumber == 6) {
                return 'P';
            }
            else if (positionNumber == 7) {
                return 'r';
            }
            else if (positionNumber == 8) {
                return 'n';
            }
            else if (positionNumber == 9) {
                return 'b';
            }
            else if (positionNumber == 10) {
                return 'q';
            }
            else if (positionNumber == 11) {
                return 'k';
            }
            else if (positionNumber == 12) {
                return 'p';
            }
            else {
                return '!';
            }
        }

        void printBoard() {
            int tmp;
            char output[17];
            output[16] = '\0';
            for (int row = 7; row >= 0; row--) {
                for (int col = 0; col < 16; col += 2) {
                    tmp = board[row][col / 2];
                    output[col] = getPositionAsChar(tmp);
                    output[col + 1] = '|';
                }
                printf("%s", output);
                printf("\n----------------\n");
            }
            printf("\n\n");
        }

        void printAllPossibleBoards() {
            for (int boardNo = 0; boardNo < movesChecked; boardNo++) {
                possibleMoves[boardNo]->printBoard();
            }
        }
    };

    class GPU_Functions {
    private:
        static float* memTransferArray(float* a, int a_rows, int a_cols, float* dev_a){
            //hipMalloc((void**)&dev_a, sizeof(float)*);
        }
        
        static void ()
    public:
        static void matMult(float** a, float** b, float** c, int a_rows, int a_cols, int b_cols) {
            float* dev_a = 0;
            float* dev_b = 0;
            float* dev_c = 0;

            float* host_a
            //memTransferArray(a);
            
        }
    };


    class Layer {
    private:
        int neuronsCurrentLayer = 0;
        int neuronsNextLayer = 0;
        int numberOfWeights = 0;
        float** weightsArray;
        float* biasArray;
        float* outputArray;

        //virtual void computeLayer() = 0;

        void swishActivationFunction() {

        }

    public:
        Layer(int neuronsCurrentLayer) {
            this->neuronsCurrentLayer = neuronsCurrentLayer;
        }

        void calculateOutput(int* inputArr) {

        }

        void generateArrays(int neuronsNextLayer) {
            this->neuronsNextLayer = neuronsNextLayer;

            //Allocate Memory for weightArray
            this->numberOfWeights = neuronsCurrentLayer * neuronsNextLayer;
            weightsArray = (float**)malloc(sizeof(float*) * neuronsCurrentLayer);
            for (int i = 0; i < neuronsNextLayer; i++) {
                weightsArray[i] = (float*)malloc(sizeof(float) * neuronsNextLayer);
            }

            //Allocate Memory for biasArray
            biasArray = (float*)malloc(sizeof(float) * neuronsNextLayer);

            //Initialise weights and bias to 0
            for (int j = 0; j < neuronsNextLayer; j++) {
                for (int i = 0; i < neuronsCurrentLayer; i++){
                    weightsArray[i][j] = 0;
                }
                biasArray[j] = 0;
            }
        }

        int getNeuronCount() {
            return neuronsCurrentLayer;
        }

        void printWeightsArray() {
            for (int i = 0; i < neuronsCurrentLayer; i++) {
                for (int j = 0; j < neuronsNextLayer; j++) {
                    printf("%d\t", weightsArray[i][j]);
                }
                printf("\n");
            }
            printf("\n");
        }

        void printBiasArray() {
            for (int i = 0; i < neuronsNextLayer; i++) {
                printf("%d\n", biasArray[i]);
            }
            printf("\n");
        }
};


    class FullyConnectedLayer : public Layer {
    public:
        FullyConnectedLayer(int neurons) : Layer(neurons){}
    };

    class NeuralNetwork {
    private:
        int numberOfLayers = 0;
        Layer** layersArray = (Layer**)malloc(sizeof(Layer));


    public:
        void addLayer(Layer* newLayer) {
            if (numberOfLayers == 0) {\
                numberOfLayers++;
                layersArray[0] = newLayer;
            }
            else {
                //Resize the layers array
                Layer** tmpArray = (Layer**)malloc(sizeof(Layer)*numberOfLayers);
                for (int i = 0; i < numberOfLayers; i++) {
                    tmpArray[i] = layersArray[i];
                }

                free(layersArray);
                numberOfLayers++;
                layersArray = (Layer**)malloc(sizeof(layersArray) * numberOfLayers);

                for (int i = 0; i < numberOfLayers-1; i++) {
                     layersArray[i] = tmpArray[i];
                }

                //Add new layer
                layersArray[numberOfLayers-1] = newLayer;

                //Generate weights for previous layer
                layersArray[numberOfLayers - 2]->generateArrays(newLayer->getNeuronCount());
            }
        }

        void compute(int* arr) {

        }

        void printLayerWeights(int layerNumber) {
            this->layersArray[layerNumber]->printWeightsArray();
        }

        void printLayerBias(int layerNumber) {
            this->layersArray[layerNumber]->printBiasArray();
        }
    };

    int main()
    {

        //Board* board = new Board();
        //board->getMoves(true);
        //board->printAllPossibleBoards();
        //NeuralNetwork* test = new NeuralNetwork();
        //test->addLayer(new FullyConnectedLayer(5));
        //test->addLayer(new FullyConnectedLayer(10));
        //test->printLayerBias(0);
        //test->printLayerWeights(0);
        

        return 0;
    }
