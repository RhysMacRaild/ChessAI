
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>

class Moves {
private:
    int board[8][8];

public:
    Moves() {
        for (int row; row = 0; row++) {
            for (int col; col = 0; col++) {
                board[row][col] = 0;
            }
        }
    }
};

class Board {
private:
    int turn = 0;
    int movesChecked = 0; 
    Board *possibleMoves[1000];
    Board *gameHistory[300];



    //Move a piece and create a new board
    void movePiece(Board* currentBoard, int initialRow, int initialCol, int movedTooRow, int movedTooCol) {
        for (int row = 0; row < 8; row++) {
            for (int col = 0; col < 8; col++) {
                //Remove the piece from its inital square
                if (row == initialRow && col == initialCol) {
                    board[row][col] = 0;
                }
                //Move the piece to new square
                else if (row == movedTooRow && col == movedTooCol) {
                    board[row][col] = currentBoard->board[initialRow][initialCol];
                }

                //Copy the rest of the board
                else {
                    board[row][col] = currentBoard->board[row][col];
                    //(board[row]) + col = (currentBoard->board[row]) + col;
                }
            }
        }
    }

    bool bishopCheck(int initRow, int initCol, int newRow, int newCol, bool white) {
        //Toggle between checking valid white and black moves
        int lowerBound;
        int upperBound; 
        if (!white) {
            lowerBound = 1;
            upperBound = 6;
        }
        else {
            lowerBound = 7;
            upperBound = 12;
        }
        
        if (newCol >= 0 && newCol < 8) {
            if (board[newRow][newCol] == 0) {
                Board* move1 = new Board(this, initRow, initCol, newRow, newCol);
                return true;
            }
            else if (board[newRow][newCol] >= lowerBound && board[newRow][newCol] <= upperBound) {
                Board* move1 = new Board(this, initRow, initCol, newRow, newCol);
                return false;
            }

            else {
                return false;
            }
        }
    }

public:
    int board[8][8] = { 
        {1,2,3,4,5,3,2,1},
        {6,6,6,6,6,6,6,6},
        {0,0,0,0,0,0,0,0},
        {0,0,0,0,0,0,0,0},
        {0,0,0,0,0,0,0,0},
        {0,0,0,0,0,0,0,0},
        {12,12,12,12,12,12,12,12},
        {7,8,9,10,11,9,8,7}};

    Board() {
        printBoard();
    }

    //Validate move and generate new board if valid
    Board(Board* currentBoard, int initialRow, int initialCol, int movedTooRow, int movedTooCol) {
        //Check if proposed move exists
        if ((movedTooRow >= 8 || movedTooRow < 0) || (movedTooCol >= 8 || movedTooCol < 0)) {
            delete this;
        }
        else {
            //Check if move is possible for white
            if (currentBoard->board[initialRow][initialCol] < 7) {
                //Check if a white piece is trying to move to the position of another white piece
                if (currentBoard->board[movedTooRow][movedTooCol] < 7 && currentBoard->board[movedTooRow][movedTooCol] > 0) {
                    delete this;
                }
                else {
                    movePiece(currentBoard, initialRow, initialCol, movedTooRow, movedTooCol);
                    currentBoard->addBoardToPossible(this);
                }
            }
            //Check for black
            else {
                //Check if black piece if moving to the position of another black piece
                if (currentBoard->board[movedTooRow][movedTooCol] >= 7 && currentBoard->board[movedTooRow][movedTooCol] <= 12) {
                    delete this;
                }
                else {
                    movePiece(currentBoard, initialRow, initialCol, movedTooRow, movedTooCol);
                    currentBoard->addBoardToPossible(this);
                }
            }
        }
    }

    void getMoves(bool white) {
        //White Moves
        if (white) {
            for (int row = 0; row < 8; row++) {
                for (int col = 0; col < 8; col++) {
                    //Pawn
                    if (getPositionAsChar(board[row][col]) == 'P' && row == 1) {
                        Board* move1 = new Board(this, row, col, row + 1, col);
                        Board* move2 = new Board(this, row, col, row + 2, col);
                    }
                    else if (getPositionAsChar(board[row][col]) == 'P') {
                        Board* move1 = new Board(this, row, col, row + 1, col);
                    }

                    //Knight
                    else if (getPositionAsChar(board[row][col]) == 'N') {
                        Board* move1 = new Board(this, row, col, row + 2, col - 1);
                        Board* move2 = new Board(this, row, col, row + 2, col + 1);
                        Board* move3 = new Board(this, row, col, row - 1, col + 2);
                        Board* move4 = new Board(this, row, col, row + 1, col + 2);
                        Board* move5 = new Board(this, row, col, row - 2, col - 1);
                        Board* move6 = new Board(this, row, col, row - 2, col + 1);
                        Board* move7 = new Board(this, row, col, row - 1, col - 2);
                        Board* move8 = new Board(this, row, col, row + 1, col - 2);
                    }

                    //Bishop
                    else if (getPositionAsChar(board[row][col]) == 'B') {
                        bool upRight = true;
                        bool upLeft = true;
                        bool downRight = true;
                        bool downLeft = true;

                        //Check rows above current
                        for (int newRow = row+1; newRow < 8; newRow++) {

                            int moveUpRightCol = col + (newRow - row);
                            int moveUpLeftCol = col - (newRow - row);

                            //UpRight
                            if (upRight) {
                                upRight = bishopCheck(row, col, newRow, moveUpRightCol, white);
                            }

                            if (upLeft) {
                                upLeft = bishopCheck(row, col, newRow, moveUpLeftCol, white);
                            }

                        }

                        //Check rows below current
                        for (int newRow = row - 1; newRow >=0 ; newRow--) {

                            int moveDownRightCol = col + (row - newRow);
                            int moveDownLeftCol = col - (row - newRow);

                            //UpRight
                            if (downRight) {
                                downRight = bishopCheck(row, col, newRow, moveDownRightCol, white);
                            }

                            if (downLeft) {
                                downLeft = bishopCheck(row, col, newRow, moveDownLeftCol, white);
                            }

                        }

                    }
                }
            }
        }
        //Black moves
        else {
            for (int row = 0; row < 8; row++) {
                for (int col = 0; col < 8; col++) {
                    //Pawn
                    if (getPositionAsChar(board[row][col]) == 'p' && row == 6) {
                        Board* move1 = new Board(this, row, col, row - 1, col);
                        Board* move2 = new Board(this, row, col, row - 2, col);
                    }
                    else if (getPositionAsChar(board[row][col]) == 'p') {
                        Board* move1 = new Board(this, row, col, row - 1, col);
                    }

                    //Knight
                    else if (getPositionAsChar(board[row][col]) == 'n') {
                        Board* move1 = new Board(this, row, col, row + 2, col - 1);
                        Board* move2 = new Board(this, row, col, row + 2, col + 1);
                        Board* move3 = new Board(this, row, col, row - 1, col + 2);
                        Board* move4 = new Board(this, row, col, row + 1, col + 2);
                        Board* move5 = new Board(this, row, col, row - 2, col - 1);
                        Board* move6 = new Board(this, row, col, row - 2, col + 1);
                        Board* move7 = new Board(this, row, col, row - 1, col - 2);
                        Board* move8 = new Board(this, row, col, row + 1, col - 2);
                    }
                }
            }
        }
    }
    
    void addBoardToPossible(Board *possibleBoard) {
        for (int row = 0; row < 8; row++) {
            for (int col = 0; col < 8; col++) {
                this->possibleMoves[movesChecked] = possibleBoard;
            }
        }
        movesChecked++;
    }

    void submitTurn(Board* newBoard) {
        for (int row = 0; row < 8; row++) {
            for (int col = 0; col < 8; col++) {
                this->gameHistory[turn]->board[row][col] = this->board[row][col];
                this->board[row][col] = newBoard->board[row][col];
            }
        }
        turn++;
    }

    char getPositionAsChar(int positionNumber) {
        if (positionNumber == 0) {
            return ' ';
        }
        else if (positionNumber == 1) {
            return 'R';
        }
        else if (positionNumber == 2) {
            return 'N';
        }
        else if (positionNumber == 3) {
            return 'B';
        }
        else if (positionNumber == 4) {
            return 'Q';
        }
        else if (positionNumber == 5) {
            return 'K';
        }
        else if (positionNumber == 6) {
            return 'P';
        }
        else if (positionNumber == 7) {
            return 'r';
        }
        else if (positionNumber == 8) {
            return 'n';
        }
        else if (positionNumber == 9) {
            return 'b';
        }
        else if (positionNumber == 10) {
            return 'q';
        }
        else if (positionNumber == 11) {
            return 'k';
        }
        else if (positionNumber == 12) {
            return 'p';
        }
        else {
            return '!';
        }
    }

    void printBoard() {
        int tmp;
        char output[17];
        output[16] = '\0';
        for (int row = 7; row >= 0; row--) {
            for (int col = 0; col < 16; col+=2) {
                tmp = board[row][col/2];
                output[col] = getPositionAsChar(tmp);
                output[col + 1] = '|';
            }
            printf("%s", output);
            printf("\n----------------\n");
        }
        printf("\n\n");
    }

    void printAllPossibleBoards() {
        for (int boardNo = 0; boardNo < movesChecked; boardNo++) {
            possibleMoves[boardNo]->printBoard();
        }
    }
};


int main()
{
    Board* board = new Board();
    board->getMoves(true);
    board->printAllPossibleBoards();

    return 0;
}

